
#include <hip/hip_runtime.h>
#include <stdint.h>

// Left rotation
__device__ uint32_t rotl(uint32_t x, int n) {
    return (x << n) | (x >> (32 - n));
}

__device__ void ripemd160(const uint8_t *msg, uint8_t *digest) {
    uint32_t h0 = 0x67452301, h1 = 0xefcdab89, h2 = 0x98badcfe;
    uint32_t h3 = 0x10325476, h4 = 0xc3d2e1f0;

    uint8_t block[64] = {0};
    for (int i = 0; i < 32; i++) block[i] = msg[i];
    block[32] = 0x80;
    block[56] = 32 * 8;

    uint32_t X[16];
    for (int i = 0; i < 16; i++) {
        X[i] = (uint32_t)block[i*4] |
               ((uint32_t)block[i*4+1] << 8) |
               ((uint32_t)block[i*4+2] << 16) |
               ((uint32_t)block[i*4+3] << 24);
    }

    uint32_t A = h0, B = h1, C = h2, D = h3, E = h4;
    uint32_t AA = h0, BB = h1, CC = h2, DD = h3, EE = h4;

    const int r1[80] = {
         0, 1, 2, 3, 4, 5, 6, 7, 8, 9,10,11,12,13,14,15,
         7, 4,13, 1,10, 6,15, 3,12, 0, 9, 5, 2,14,11, 8,
         3,10,14, 4, 9,15, 8, 1, 2, 7, 0, 6,13,11, 5,12,
         1, 9,11,10, 0, 8,12, 4,13, 3, 7,15,14, 5, 6, 2,
         4, 0, 5, 9, 7,12, 2,10,14, 1, 3, 8,11, 6,15,13
    };
    const int r2[80] = {
         5,14, 7, 0, 9, 2,11, 4,13, 6,15, 8, 1,10, 3,12,
         6,11, 3, 7, 0,13, 5,10,14,15, 8,12, 4, 9, 1, 2,
        15, 5, 1, 3, 7,14, 6, 9,11, 8,12, 2,10, 0,13, 4,
         8, 6, 4, 1, 3,11,15, 0, 5,12, 2,13, 9, 7,10,14,
        12,15,10, 4, 1, 5, 8, 7, 6, 2,13,14, 0, 3, 9,11
    };
    const int s1[80] = {
        11,14,15,12, 5, 8, 7, 9,11,13,14,15, 6, 7, 9, 8,
         7, 6, 8,13,11, 9, 7,15, 7,12,15, 9,11, 7,13,12,
        11,13, 6, 7,14, 9,13,15,14, 8,13, 6, 5,12, 7, 5,
        11,12,14,15,14,15, 9, 8, 9,14, 5, 6, 8, 6, 5,12,
         9,15, 5,11, 6, 8,13,12, 5,12,13,14,11, 8, 5, 6
    };
    const int s2[80] = {
         8, 9, 9,11,13,15,15, 5, 7, 7, 8,11,14,14,12, 6,
         9,13,15, 7,12, 8, 9,11, 7, 7,12, 7, 6,15,13,11,
         9, 7,15,11, 8, 6, 6,14,12,13, 5,14,13,13, 7, 5,
        15, 5, 8,11,14,14, 6,14, 6, 9,12, 9,12, 5,15, 8,
         8, 5,12, 9,12, 5,14, 6, 8,13, 6, 5,15,13,11,11
    };
    const uint32_t K1[5] = {0x00000000,0x5a827999,0x6ed9eba1,0x8f1bbcdc,0xa953fd4e};
    const uint32_t K2[5] = {0x50a28be6,0x5c4dd124,0x6d703ef3,0x7a6d76e9,0x00000000};

    for (int j = 0; j < 80; j++) {
        int round = j / 16;
        uint32_t f = (round == 0) ? (B ^ C ^ D) :
                     (round == 1) ? ((B & C) | (~B & D)) :
                     (round == 2) ? ((B | ~C) ^ D) :
                     (round == 3) ? ((B & D) | (C & ~D)) :
                                    (B ^ (C | ~D));

        uint32_t T = rotl(A + f + X[r1[j]] + K1[round], s1[j]) + E;
        A = E; E = D; D = rotl(C, 10); C = B; B = T;

        f = (round == 0) ? (BB ^ (CC | ~DD)) :
            (round == 1) ? ((BB & DD) | (CC & ~DD)) :
            (round == 2) ? ((BB | ~CC) ^ DD) :
            (round == 3) ? ((BB & CC) | (~BB & DD)) :
                           (BB ^ CC ^ DD);

        T = rotl(AA + f + X[r2[j]] + K2[round], s2[j]) + EE;
        AA = EE; EE = DD; DD = rotl(CC, 10); CC = BB; BB = T;
    }

    uint32_t tmp = h1 + C + DD;
    h1 = h2 + D + EE;
    h2 = h3 + E + AA;
    h3 = h4 + A + BB;
    h4 = h0 + B + CC;
    h0 = tmp;

    digest[ 0] = h0 & 0xff;   digest[ 1] = (h0 >> 8) & 0xff;
    digest[ 2] = (h0 >> 16) & 0xff; digest[ 3] = (h0 >> 24) & 0xff;
    digest[ 4] = h1 & 0xff;   digest[ 5] = (h1 >> 8) & 0xff;
    digest[ 6] = (h1 >> 16) & 0xff; digest[ 7] = (h1 >> 24) & 0xff;
    digest[ 8] = h2 & 0xff;   digest[ 9] = (h2 >> 8) & 0xff;
    digest[10] = (h2 >> 16) & 0xff; digest[11] = (h2 >> 24) & 0xff;
    digest[12] = h3 & 0xff;   digest[13] = (h3 >> 8) & 0xff;
    digest[14] = (h3 >> 16) & 0xff; digest[15] = (h3 >> 24) & 0xff;
    digest[16] = h4 & 0xff;   digest[17] = (h4 >> 8) & 0xff;
    digest[18] = (h4 >> 16) & 0xff; digest[19] = (h4 >> 24) & 0xff;
}

extern "C" __global__ void ripemd160_gpu(const uint8_t *sha256_hashes, uint8_t *ripemd160_out, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;
    ripemd160(&sha256_hashes[idx * 32], &ripemd160_out[idx * 20]);
}

